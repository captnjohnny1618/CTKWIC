#include "hip/hip_runtime.h"
/* CTBangBang is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <regex.h>
#include <cstdarg>
#include <unistd.h>
#include <sys/types.h>
#include <pwd.h>

#include <recon_structs.h>
#include <setup.h>
#include <preprocessing.h>
#include <rebin_filter.h>
#include <rebin_filter_cpu.h>
#include <backproject.h>
#include <backproject_cpu.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
	{
	    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	    if (abort) exit(code);
	}
}

void log(int verbosity, const char *string, ...);
void split_path_file(char**p, char**f, char *pf);
void write_float(float * array,size_t numel,const char * file){

    FILE * fid=fopen(file,"w");
    fwrite(array,sizeof(float),numel,fid);
    fclose(fid);

}


void usage(){
    printf("\n");
    printf("usage: recon [options] input_prm_file\n\n");
    printf("    Options:\n");
    printf("          -v: verbose.\n");
    printf("          -t: test files will be written to desktop.\n");
    printf("    --no-gpu: run program exclusively on CPU. Will override --device=i option.\n");
    printf("  --device=i: run on GPU device number 'i'\n");
    printf("    --timing: Display timing information for each step of the recon process\n");
    printf(" --benchmark: Writes timing data to file used by benchmarking tool\n");
    printf("\n");
    printf("Copyright John Hoffman 2015\n\n");
    exit(0);
}


int main(int argc, char ** argv){

    struct recon_metadata mr;
    memset(&mr,0,sizeof(struct recon_metadata));

    // Parse any command line arguments
    if (argc<2)
	usage();
    
    regex_t regex_dev;
    regmatch_t regmatch_dev;
    if (regcomp(&regex_dev,"--device=*",0)!=0){
	printf("Regex didn't work properly\n");
	exit(1);
    }
    
    for (int i=1;i<(argc-1);i++){
	if (strcmp(argv[i],"-t")==0){
	    mr.flags.testing=1;
	}
	else if (strcmp(argv[i],"-v")==0){
	    mr.flags.verbose=1;
	}
	else if (strcmp(argv[i],"--no-gpu")==0){
	    mr.flags.no_gpu=1;
	}
	else if (regexec(&regex_dev,argv[i],1,&regmatch_dev,0)==0){
	    mr.flags.set_device=1;
	    sscanf(argv[i],"--device=%d",&mr.flags.device_number);
	}
	else if (strcmp(argv[i],"--timing")==0){
	    mr.flags.timing=1;
	}
	else if (strcmp(argv[i],"--benchmark")==0){
	    mr.flags.benchmark=1;
	}
	else{
	    usage();
	}
    }

    log(mr.flags.verbose,"\n-----------------------------------------\n"
	"|      THIS IS NOT CTBangBang!!!!       |\n"
	"-----------------------------------------\n\n");

    log(mr.flags.verbose,"CHECKING INPUT PARAMETERS AND CONFIGURING RECONSTRUCTION\n"
	"\n");
    
    /* --- Get working directory and User's home directory --- */
    struct passwd *pw=getpwuid(getuid());
    
    const char * homedir=pw->pw_dir;
    strcpy(mr.homedir,homedir);
    char full_exe_path[4096]={0};
    char * exe_path=(char*)calloc(4096,sizeof(char));
    char * exe_name=(char*)calloc(255,sizeof(char));
    readlink("/proc/self/exe",full_exe_path,4096);
    split_path_file(&exe_path,&exe_name,full_exe_path);
    strcpy(mr.install_dir,exe_path);
    mr.install_dir[strlen(mr.install_dir)-1]=0;
    
    /* --- Step 0: configure our processor (CPU or GPU) */
    // We want to send to the GPU furthest back in the list which is
    // unlikely to have a display connected.  We also check for the
    // user passing a specific device number via the command line

    int device_count=0;
    hipGetDeviceCount(&device_count);
    if (device_count==0){
	mr.flags.no_gpu=1;
    }

    // Configure the GPU/CPU selection
    if (mr.flags.no_gpu==0){
	int device;
	if (mr.flags.set_device==1){
	    log(mr.flags.verbose,"CUDA device %d requested.\n",mr.flags.device_number);
	    log(mr.flags.verbose,"Attempting to set device... ");
	    hipSetDevice(mr.flags.device_number);
	    hipGetDevice(&device);
	    if (device!=mr.flags.device_number){
		printf("There was a problem setting device.\n");
	    }
	    else{
		log(mr.flags.verbose,"success!\n");
	    }
	}
	else{
	    hipSetDevice(device_count-1);
	    hipGetDevice(&device);
	}	
	log(mr.flags.verbose,"Working on GPU %i \n",device);
	hipDeviceReset();
    }
    else{
	log(mr.flags.verbose,"Working on CPU\n");
    }

    // --timing cuda events
    hipEvent_t start,stop;

    hipEvent_t bench_master_start,bench_master_stop,bench_start,bench_stop;
    if (mr.flags.benchmark){
	hipEventCreate(&bench_master_start);
	hipEventCreate(&bench_master_stop);
	hipEventRecord(bench_master_start);
    }
    
    /* --- Step 1-3 handled by functions in setup.cu --- */
    // Step 1: Parse input file
    log(mr.flags.verbose,"Reading PRM file...\n");
    mr.rp=configure_recon_params(argv[argc-1]);

    /* --- Check for defined output directory, set to desktop if empty --- */
    strcpy(mr.output_dir,mr.rp.output_dir);
    if (strcmp(mr.output_dir,"")==0){
	char fullpath[4096+255];
	strcpy(fullpath,mr.homedir);
	strcat(fullpath,"/Desktop/");
	strcpy(mr.output_dir,fullpath);
    }
    
    // Set up benchmarking variables and output file if requested
    char fullpath[4096+255];
    strcpy(fullpath,mr.output_dir);
    strcat(fullpath,".tmp_benchmark.bin");
    FILE * benchmark_file;
    if (mr.flags.benchmark){
	benchmark_file=fopen(fullpath,"a");
	fseek(benchmark_file,0,SEEK_END);
    }

    // Step 2a: Setup scanner geometry
    log(mr.flags.verbose,"Configuring scanner geometry...\n");
    mr.cg=configure_ct_geom(&mr);
    
    // Step 2b: Configure all remaining information
    log(mr.flags.verbose,"Configuring final reconstruction parameters...\n");
    configure_reconstruction(&mr);

    log(mr.flags.verbose,"Allowed recon range: %.2f to %.2f\n",mr.ri.allowed_begin,mr.ri.allowed_end);

    log(mr.flags.verbose,"\nSTARTING RECONSTRUCTION\n\n");


    size_t n_proj_rebin_chunk=5000;
    size_t kwic_blocks=5;
    //size_t kwic_blocks=mr.rp.n_readings/n_proj_rebin_chunk;
 

    
    for (int i=0;i<kwic_blocks;i++){

	/* override idx_pull_start and n_proj_pull after each block update */
	mr.ri.idx_pull_start=i*n_proj_rebin_chunk;
	mr.ri.idx_pull_end=(i+1)*n_proj_rebin_chunk;

	update_block_info(&mr);
	printf("%lu\n",mr.ri.idx_pull_start);
	printf("%lu\n",mr.ri.idx_pull_end);
	printf("%lu\n",mr.ri.n_proj_pull);
	
	log(mr.flags.verbose,"----------------------------\n"
	    "Working on block %d of %d \n",i+1,kwic_blocks);
	
	// Step 3: Extract raw data from file into memory
	log(mr.flags.verbose,"Reading raw data from file...\n");
	extract_projections(&mr);

	/* --- Step 3.5: Adaptive filtration handled by preprocessing.cu ---*/
	// Step 3.5: Adaptive filtration of raw data to reduce streak artifacts
	log(mr.flags.verbose,"Running adaptive filtering...\n");

	if (mr.flags.timing){
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
	}

	adaptive_filter_kk(&mr);

	if (mr.flags.timing){
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,start,stop);
	    printf("%.2f ms for adaptive filtration\n",milli);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	}

	/* --- Step 4 handled by functions in rebin_filter.cu --- */
	// Step 4: Rebin and filter
	log(mr.flags.verbose,"Rebinning and filtering data...\n");

	if (mr.flags.timing){
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
	}

	if (mr.flags.benchmark){
	    hipEventCreate(&bench_start);
	    hipEventCreate(&bench_stop);
	    hipEventRecord(bench_start);
	}

	printf("%.2f\n",mr.cg.central_channel);
	
	if (mr.flags.no_gpu==1)
	    rebin_filter_cpu(&mr);
	else
	    rebin_filter(&mr);

	printf("%.2f\n",mr.cg.central_channel);
	
	if (mr.flags.timing){
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,start,stop);
	    printf("%.2f ms to rebin & filter\n",milli);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	}
	if (mr.flags.benchmark){
	    hipEventRecord(bench_stop);
	    hipEventSynchronize(bench_stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,bench_start,bench_stop);
	    // write the benchmark data to file
	    fwrite(&milli,sizeof(float),1,benchmark_file);
	    hipEventDestroy(bench_start);
	    hipEventDestroy(bench_stop);
	}

	// Copy rebin chunk over to full rebin_array
	size_t rebin_idx_start=mr.ri.idx_pull_start/mr.ri.n_ffs;
	size_t rebin_idx_end=mr.ri.idx_pull_end/mr.ri.n_ffs;
	size_t rebin_size=mr.cg.n_channels_oversampled*mr.cg.n_rows*mr.ri.n_proj_pull/mr.ri.n_ffs;
	memcpy(&mr.ctd.full_rebin[i*n_proj_rebin_chunk/mr.ri.n_ffs*mr.cg.n_channels_oversampled*mr.cg.n_rows],mr.ctd.rebin,rebin_size*sizeof(float));
	write_float(mr.ctd.rebin,rebin_size,"/home/john/Desktop/rebin_test.bin");
    }

    // Write the Full rebin out to disk
    //write_float(mr.ctd.full_rebin,mr.cg.n_channels_oversampled*mr.cg.n_rows*mr.rp.n_readings/mr.ri.n_ffs,"/home/john/Desktop/rebinned_raw.bin");
    write_float(mr.ctd.full_rebin,mr.cg.n_channels_oversampled*mr.cg.n_rows*kwic_blocks*n_proj_rebin_chunk/mr.ri.n_ffs,"/home/john/Desktop/rebinned_raw.bin");
    write_float(mr.tube_angles,kwic_blocks*n_proj_rebin_chunk/mr.ri.n_ffs,"/home/john/Desktop/tube_angles.bin");    
    
    // Step 6: Save image data to disk (found in setup.cu)
    log(mr.flags.verbose,"----------------------------\n\n");
    log(mr.flags.verbose,"Writing image data to %s%s.img\n",mr.output_dir,mr.rp.raw_data_file);
    finish_and_cleanup(&mr);

    log(mr.flags.verbose,"Done.\n");

    if (mr.flags.benchmark){
	hipEventRecord(bench_master_stop);
	hipEventSynchronize(bench_master_stop);
	float milli=0.0f;
	hipEventElapsedTime(&milli,bench_master_start,bench_master_stop);
	// write the benchmark data to file
	fwrite(&milli,sizeof(float),1,benchmark_file);
	hipEventDestroy(bench_master_start);
	hipEventDestroy(bench_master_stop);
	fclose(benchmark_file);
    }

    hipDeviceReset();
    return 0;
   
}

void log(int verbosity, const char *string,...){
    va_list args;
    va_start(args,string);

    if (verbosity){
	vprintf(string,args);
	va_end(args);
    } 
}

void split_path_file(char**p, char**f, char *pf) {
    char *slash = pf, *next;
    while ((next = strpbrk(slash + 1, "\\/"))) slash = next;
    if (pf != slash) slash++;
    *p = strndup(pf, slash - pf);
    *f = strdup(slash);
}
